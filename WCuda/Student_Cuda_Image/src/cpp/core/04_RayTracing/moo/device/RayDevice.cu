#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "cudaTools.h"
#include "Device.h"

#include "NbSpheres.h"
#include "IndiceTools_GPU.h"

#include "../../../04_RayTracing/moo/device/math/RayMath.h"
using namespace gpu;

// Attention : 	Choix du nom est impotant!
//		VagueDevice.cu et non Vague.cu
// 		Dans ce dernier cas, probl�me de linkage, car le nom du .cu est le meme que le nom d'un .cpp (host)
//		On a donc ajouter Device (ou n'importequoi) pour que les noms soient diff�rents!

/*----------------------------------------------------------------------*\
 |*			Declaration 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Imported	 	*|
 \*-------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__global__ void rayGM(float4* ptrDevPixels, int nbSphere, Sphere* ptrDevTabSphere, uint w, uint h, float t);
__global__ void rayCM(float4* ptrDevPixels, uint w, uint h, float t);
__global__ void raySM(float4* ptrDevPixels, int nbSphere, Sphere* ptrDevTabSphere, uint w, uint h, float t);

__device__ void work(float4* ptrDevPixels, int nbSphere, Sphere* ptrDevTabSphere, uint w, uint h, float t);
__device__ void copyGMtoSM(Sphere* ptrTabSphere, Sphere* ptrTabSphreSM, int n);

__host__ void uploadGPU(Sphere* ptrDevtabValue);

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

/*----------------------------------------------------------------------*\
 |*			Implementation 					*|
 \*---------------------------------------------------------------------*/

/*--------------------------------------*\
 |*		Public			*|
 \*-------------------------------------*/

__host__ void uploadGPU(Sphere* ptrDevtabValue)
    {
    size_t size = NB_SPHERE * sizeof(Sphere);
    int offset=0;

    HANDLE_ERROR (hipMemcpyToSymbol(HIP_SYMBOL(
	    TAB_CM),ptrDevtabValue,
	    size,
	    offset,
	    hipMemcpyHostToDevice));
    }

__global__ void rayGM(float4* ptrDevPixels, int nbSphere, Sphere* ptrDevTabSphere, uint w, uint h, float t)
    {
    work(ptrDevPixels, nbSphere, ptrDevTabSphere, w, h, t);
    }

__global__ void rayCM(float4* ptrDevPixels, uint w, uint h, float t)
    {
    work(ptrDevPixels, NB_SPHERE, TAB_CM, w, h, t);
    }

__global__ void raySM(float4* ptrDevPixels, int nbSphere, Sphere* ptrDevTabSphere, uint w, uint h, float t)
    {
    __shared__ extern Sphere tabSM[];
    copyGMtoSM(ptrDevTabSphere, tabSM, nbSphere);
    __syncthreads();
    work(ptrDevPixels, nbSphere, tabSM, w, h, t);
    }

__device__ void copyGMtoSM(Sphere* ptrTabSphere, Sphere* ptrTabSphreSM, int n)
    {
    const int TID_LOCAL = Indice2D::tidLocal();
    const int NB_THREAD_LOCAL = Indice2D::nbThreadLocal();
    int s = TID_LOCAL;
    while (s < n)
	{
	ptrTabSphreSM[s] = ptrTabSphere[s];
	s += NB_THREAD_LOCAL;
	}
    }

/*--------------------------------------*\
 |*		Private			*|
 \*-------------------------------------*/

__device__ void work(float4* ptrDevPixels, int nbSphere, Sphere* ptrDevTabSphere, uint w, uint h, float t)
    {
	RayMath rayMath = RayMath(nbSphere, ptrDevTabSphere);

        const int TID = Indice2D::tid();
        const int NB_THREAD = Indice2D::nbThread();
        const int WH = w * h;

        int i;	// in [0,h[
        int j; 	// in [0,w[

        int s = TID; // in [0,...

        while (s < WH)
    	{
    	IndiceTools::toIJ(s, w, &i, &j); // s[0,W*H[ --> i[0,H[ j[0,W[

    	rayMath.colorIJ(&ptrDevPixels[s], i, j, t); // update ptrTabPixels[s]

    	s += NB_THREAD;
    	}
    }

/*----------------------------------------------------------------------*\
 |*			End	 					*|
 \*---------------------------------------------------------------------*/

