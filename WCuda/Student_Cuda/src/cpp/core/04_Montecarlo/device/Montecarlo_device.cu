#include "hip/hip_runtime.h"
#include "Indice2D.h"
#include "Indice1D.h"
#include "cudaTools.h"
#include <hiprand/hiprand_kernel.h>
#include "reductionADD.h"
#include <stdio.h>


__global__ void montecarlo(uint* p_ary_gm, hiprandState* ptr_ary_gen_gm, uint n);

__device__ float f(float x);

__device__ void useGenerator(hiprandState* ptr_ary_gen_gm, uint n, uint tabSM[]);

__device__ float f(float x){
    return 1. / (1 + x * x);
    }

__global__ void montecarlo(uint* p_ary_gm, hiprandState* ptr_ary_gen_gm, uint n){
    extern __shared__ uint tabSM[];
    useGenerator(ptr_ary_gen_gm, n, tabSM);
    __syncthreads();
    reductionADD<uint>(tabSM, p_ary_gm);
    }

__global__ void createGenerator(hiprandState* ptr_ary_gen_gm, int deviceId)
    {
    const int TID = Indice1D::tid();
    int deltaSeed = deviceId * INT_MAX / 10000;
    int deltaSequence = deviceId * 100;
    int deltaOffset = deviceId * 100;
    int seed = 1337 + deltaSeed;
    int sequenceNumber = TID + deltaSequence;
    int offset = deltaOffset;

    hiprand_init(seed, sequenceNumber, offset, &ptr_ary_gen_gm[TID]);
    }

__device__ void useGenerator(hiprandState* ptr_ary_gen_gm, uint n, uint tabSM[])
    {
    const int TID = Indice1D::tid();
    const int TID_LOCAL = Indice1D::tidLocal();

    hiprandState localGenerator = ptr_ary_gen_gm[TID];
    float xAlea;
    float yAlea;

    uint nx = 0;
    for (long i = 1; i <= n; i++)
	{
	xAlea = hiprand_uniform(&localGenerator);
	yAlea = hiprand_uniform(&localGenerator);

	if (yAlea <= f(xAlea)){
	    nx++;
	    }
	}

    tabSM[TID_LOCAL] = nx;
    ptr_ary_gen_gm[TID] = localGenerator;
    }

