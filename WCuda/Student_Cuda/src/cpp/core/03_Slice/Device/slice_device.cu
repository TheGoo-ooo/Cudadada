#include "hip/hip_runtime.h"
#include <iostream>
#include "cudaTools.h"
#include "Device.h"
#include "Indice1D.h"
#include "SliceMath.h"
#include "reductionADD.h"


static __device__ void intra_thread_reduction(float* shared_memory_ary, int n);

__device__ void slice_run(int n, float* d_ary) {
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();

    int s = TID;
    float dx = 1./(float)n;
    float x = 0.0;

    while(s < n){
	x = s * dx;
	d_ary[s]= SliceMath::getSlice(x);
	s += NB_THREAD;
	}
    }

__device__ void slice_run_advanced(int n, float* d_ary){
    extern __shared__ float shared_memory_ary[];
    intra_thread_reduction(shared_memory_ary, n);
    __syncthreads();
    reductionADD<float>(shared_memory_ary, d_ary);
    /*
     * Ne marche pas ..
     * /opt/api/cbi/tools/bilat_tools_cuda/303_006/INC/cudatools/header/device/reduction/reductionADD.h(147): error: no instance of overloaded function "atomicAdd" matches the argument list
     *       argument types are: (float *, float)
     *     detected during:
     *       instantiation of "void reductionInterblock(T *, T *) [with T=float]"
     *  (53): here
     * instantiation of "void reductionADD(T *, T *) [with T=float]"
     * src/cpp/core/03_Slice/Device/slice_device.cu(30): here
     */
}

__device__ void intra_thread_reduction(float* shared_memory_ary, int n){
    const int TID = Indice1D::tid();
    const int NB_THREAD = Indice1D::nbThread();
    const int LOCAL_TID = Indice1D::tidLocal();
    const float dx = 1.0f/ (float) n;

    int s = TID;
    float xs;
    float sum = 0;

    while (s < n){
	xs = s * dx;
	sum = SliceMath::getSlice(xs);
	s+= NB_THREAD;
    }
    shared_memory_ary[LOCAL_TID] = 4 * sum * dx;
}


__global__ void kernel(int n, float* d_ary) {
       slice_run(n, d_ary);
}

__global__ void kernel_a(int n, float* d_ary){
    slice_run_advanced(n, d_ary);
}
